// Matrix addition program MatrixMult.cu, Barry Wilkinson, Dec. 28, 2010.
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_matrixmult(int *gpu_a, int *gpu_b, int *gpu_c, int N) {

	int k, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

      if (col < N && row < N) {
				for (k = 0; k < N; k++) 
		        		sum += gpu_a[row * N + k] * gpu_b[k * N + col];
				gpu_c[row * N + col] = sum;
			}

}

void cpu_matrixmult(int *cpu_a, int *cpu_b, int *cpu_c, int N) {
	int row, col, k, sum;

	for (row =0; row < N; row++)   				// row of a
		for (col =0; col < N; col++) {				// column of b
			sum = 0;
			for(k = 0; k < N; k++) 
          			sum += cpu_a[row * N + k] * cpu_b[k * N + col];
			cpu_c[row * N + col] = sum;
			//d[row * N + col] = gpu_c[row *N + col];
		}
}


int main(int argc, char *argv[])  {
	int i, j; 							// loop counters
	int Grid_Dim_x=1, Grid_Dim_y=1;		//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;		//Block structure values
	int noThreads_x, noThreads_y;			// number of threads available in device, each dimension
	int noThreads_block;					// number of threads in a block
	int N = 10;  						// size of array in each dimension
	int B;
	int T;
	int *a,*b,*c,*d;
	int *dev_a, *dev_b, *dev_c;
	int size;							// number of bytes in arrays
	hipEvent_t start, stop;     				// using cuda events to measure time
	float elapsed_time_ms;       			// which is applicable for asynchronous code also
	hipEventCreate(&start);		
	hipEventCreate(&stop);
int repeat = 1;
while(repeat == 1) {
/* --------------------ENTER INPUT PARAMETERS AND ALLOCATE DATA -----------------------*/
							// keyboard input

	printf("Enter the value for N: ");
	scanf("%d", &N);
//takes in input
	int valid = 0;
	while(valid == 0) {

		printf("Enter the number of blocks: ");
		scanf("%d", &B);

		printf("Enter the number of threads: ");
		scanf("%d", &T);

		if(B > 1024 || T > 1024 || B*T < N*N) {
			printf("Invlaid input entered.");
		} else {
			valid = 1;
			Grid_Dim_x = B;
			Block_Dim_x = T;		//puts the size of blocks and thread in for the dim3
		}
	}
	
	dim3 Grid(Grid_Dim_x, Grid_Dim_x);	//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device
	size = N * N * sizeof(int);				// number of bytes in total in arrays

	a = (int*) malloc(size);					//dynamically allocated memory for arrays on host
	b = (int*) malloc(size);
	c = (int*) malloc(size);					// results from GPU
	d = (int*) malloc(size);				// results from CPU
							// load arrays with some numbers

		int row, col;
		srand(2);
		for(row=0; row < N; row++) { // load arrays with some numbers
			for(col=0; col < N; col++) {
				a[row * N + col] = rand() % 10;
				b[row * N + col] = rand() % 10; 
			}
		}

	hipMalloc((void**)&dev_a, size);			// allocate memory on device
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);

	hipEventRecord(start, 0); 			// here start time, after memcpy

	gpu_matrixmult<<<Grid,Block>>>(dev_a,dev_b,dev_c,N);
	hipMemcpy(c, dev_c, size , hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     			// measuse end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms);
	double gpuTime = elapsed_time_ms; 

/* ------------- COMPUTATION DONE ON HOST CPU ----------------------------*/

	hipEventRecord(start, 0);			// use same timing*

	cpu_matrixmult(a,b,d,N);				// do calculation on host

	hipEventRecord(stop, 0);     		// measure end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms);  // exe. time
	double cpuTime = elapsed_time_ms;

/* ------------------- check device creates correct results -----------------*/
/*
	int s;
	for(s=0;s<N*N;s++) {
		printf("%d\t", d[s]);
		if(s%N == 0 && s != 0) {
			printf("\n");
		}
	}
*/
//puts out an error is the two matricies are not the same
	printf("\n");
	int error = 0;
	int k;
	for(k=0; k<N*N; k++) {
		if(d[k] != c[k]) {
			error =1;
			break;
		} 
	}

	if(error ==1 ) 
		printf("There is an error.\n");
	else
		printf("Sequential and parallel produce the same results.\n");

	double speedupFactor;
	speedupFactor = cpuTime/gpuTime;
	printf("Speedup Factor: %lf\n", speedupFactor);

	printf("Would you like to repeat? Enter 1 for yes or 0 for no.\n");
	scanf("%d", &repeat);
}
/* --------------------- repeat program  ----------------------------------------*/
 								//  while loop to repeat calc with different parameters
/* --------------  clean up  ---------------------------------------*/
	free(a); free(b); free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}

